/*********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
float *d_values;	/* pointer to device memory */

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__device__ float do_math(float toldval, float tvalues)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
	float tnewval;
	tnewval = (2.0 * tvalues) - toldval + (sqtau *  (-2.0)*tvalues);
	return tnewval;
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(float *d_values, int tpoints, int nsteps)
{
   int i, j;
	j = (1+threadIdx.x) + blockIdx.x*32;
	if( j <= tpoints ){
		float tvalues = d_values[j];
		float toldval = tvalues;
		float tnewval;
		for(i=1; i<=nsteps; i++){
			if((j==1) || (j==tpoints))
				tnewval = 0.0;
			else
				tnewval = do_math(toldval, tvalues);
			toldval = tvalues;
			tvalues = tnewval;
		}
		d_values[j] = tvalues;
	}
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	hipMalloc(&d_values, sizeof(float)*(1+tpoints));

	check_param();
	printf("Initializing points on the line...\n");
	init_line();
	hipMemcpy(d_values, values, sizeof(float)*(1+tpoints), hipMemcpyHostToDevice);

	printf("Updating all points for all time steps...\n");
	int block;
	if(tpoints%32){
		block = 1 + tpoints/32;
		update<<<block, 32>>>(d_values, tpoints, nsteps);
	}
	else{
		block = tpoints/32;
		update<<<block, 32>>>(d_values, tpoints, nsteps);
	}
	hipMemcpy(values, d_values, sizeof(float)*(1+tpoints), hipMemcpyDeviceToHost);

	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	hipFree(d_values);
	return 0;
}
